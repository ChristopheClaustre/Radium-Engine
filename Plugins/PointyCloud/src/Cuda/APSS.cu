#include <Cuda/APSS.h>

#include <Cuda/Test.h> // <- to delete
#include <Cuda/SelectionKernel.h>
#include <Cuda/UpsamplingKernel.h>
#include <Cuda/ProjectionKernel.h>

#include <iostream>

namespace PointyCloudPlugin {
namespace Cuda {

APSS::APSS(const Vector3* positions,
           const Vector3* normals,
           const Vector4* colors,
           size_t size) :
    m_grid()
{
    m_grid = new RegularGrid(size, positions);

    // device allocation
    m_sizeOriginal = size;
    CUDA_ASSERT( hipMalloc(&m_positionOriginal, size*sizeof(Vector3)) );
    CUDA_ASSERT( hipMalloc(&m_normalOriginal,   size*sizeof(Vector3)) );
    CUDA_ASSERT( hipMalloc(&m_colorOriginal,    size*sizeof(Vector4)) );

    //TEST for test only
    // sizeFinal depends on generated splats count!
    m_sizeFinal = size;
    CUDA_ASSERT( hipMalloc(&m_positionFinal,  size*sizeof(Vector3)) );
    CUDA_ASSERT( hipMalloc(&m_normalFinal,    size*sizeof(Vector3)) );
    CUDA_ASSERT( hipMalloc(&m_colorFinal,     size*sizeof(Vector4)) );
    CUDA_ASSERT( hipMalloc(&m_splatSizeFinal, size*sizeof(Scalar)) );
    m_positionFinalHost  = new Vector3[size];
    m_normalFinalHost    = new Vector3[size];
    m_colorFinalHost     = new Vector4[size];
    m_splatSizeFinalHost = new Scalar[size];

    // other allocations ...
    // regular grid initialization ...
    // set eligibility ...

    // device transfert
    CUDA_ASSERT( hipMemcpy(m_positionOriginal, positions, size*sizeof(Vector3), hipMemcpyHostToDevice) );
    CUDA_ASSERT( hipMemcpy(m_normalOriginal,   normals,   size*sizeof(Vector3), hipMemcpyHostToDevice) );
    CUDA_ASSERT( hipMemcpy(m_colorOriginal,    colors,    size*sizeof(Vector4), hipMemcpyHostToDevice) );
}

APSS::~APSS()
{
    // device desallocation
    // ...
}

void APSS::select(/*APSS parameters*/)
{
    //TEST : copy original in final
    copy<<</*numBlocks, blockSize*/1,1>>>(m_sizeOriginal, m_positionOriginal, m_normalOriginal, m_colorOriginal,
                                          m_sizeFinal,    m_positionFinal,    m_normalFinal,    m_colorFinal, m_splatSizeFinal);
    CUDA_ASSERT( hipPeekAtLastError() );
    CUDA_ASSERT( hipDeviceSynchronize() );
}

void APSS::upsample(/*APSS parameters*/)
{
    //TODO
//    kernel<<</*numBlocks, blockSize*/>>>(...);
//    hipDeviceSynchronize();
}

void APSS::project(/*APSS parameters*/)
{
    //TODO
//    kernel<<</*numBlocks, blockSize*/>>>(...);
//    hipDeviceSynchronize();
}

void APSS::finalize()
{
    // get back final data from device to host
    CUDA_ASSERT( hipMemcpy(m_positionFinalHost,  m_positionFinal, m_sizeFinal*sizeof(Vector3), hipMemcpyDeviceToHost) );
    CUDA_ASSERT( hipMemcpy(m_normalFinalHost,    m_normalFinal, m_sizeFinal*sizeof(Vector3), hipMemcpyDeviceToHost) );
    CUDA_ASSERT( hipMemcpy(m_colorFinalHost,     m_colorFinal, m_sizeFinal*sizeof(Vector4), hipMemcpyDeviceToHost) );
    CUDA_ASSERT( hipMemcpy(m_splatSizeFinalHost, m_splatSizeFinal, m_sizeFinal*sizeof(Scalar), hipMemcpyDeviceToHost) );
}



} // namespace Cuda

} // namespace PointyCloudPlugin
