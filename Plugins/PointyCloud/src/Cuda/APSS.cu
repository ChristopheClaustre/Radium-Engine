#include <Cuda/APSS.h>

#include <Cuda/Test.h> // <- to delete
#include <Cuda/SelectionKernel.h>
#include <Cuda/UpsamplingKernel.h>
#include <Cuda/ProjectionKernel.h>

#include <thrust/device_vector.h>
#include <thrust/execution_policy.h>

#include <iostream>

namespace PointyCloudPlugin {
namespace Cuda {

APSS::APSS(const Vector3* positions,
           const Vector3* normals,
           const Vector4* colors,
           size_t size) :
    m_grid()
{
    m_grid = new RegularGrid(size, positions);

    // device allocation
    m_sizeOriginal = size;
    CUDA_ASSERT( hipMalloc(&m_positionOriginal, size*sizeof(Vector3)) );
    CUDA_ASSERT( hipMalloc(&m_normalOriginal,   size*sizeof(Vector3)) );
    CUDA_ASSERT( hipMalloc(&m_colorOriginal,    size*sizeof(Vector4)) );

    CUDA_ASSERT( hipMalloc(&m_visibility,    size*sizeof(int)) );
    CUDA_ASSERT( hipMalloc(&m_visibilitySum, size*sizeof(int)) );
    CUDA_ASSERT( hipMalloc(&m_selected,      size*sizeof(int)) );

    //TEST for test only
    // sizeFinal depends on generated splats count!
    m_sizeFinal = size;
    CUDA_ASSERT( hipMalloc(&m_positionFinal,  size*sizeof(Vector3)) );
    CUDA_ASSERT( hipMalloc(&m_normalFinal,    size*sizeof(Vector3)) );
    CUDA_ASSERT( hipMalloc(&m_colorFinal,     size*sizeof(Vector4)) );
    CUDA_ASSERT( hipMalloc(&m_splatSizeFinal, size*sizeof(Scalar)) );
    m_positionFinalHost  = new Vector3[size];
    m_normalFinalHost    = new Vector3[size];
    m_colorFinalHost     = new Vector4[size];
    m_splatSizeFinalHost = new Scalar[size];

    // other allocations ...
    // regular grid initialization ...
    // set eligibility ...

    // device transfert
    CUDA_ASSERT( hipMemcpy(m_positionOriginal, positions, size*sizeof(Vector3), hipMemcpyHostToDevice) );
    CUDA_ASSERT( hipMemcpy(m_normalOriginal,   normals,   size*sizeof(Vector3), hipMemcpyHostToDevice) );
    CUDA_ASSERT( hipMemcpy(m_colorOriginal,    colors,    size*sizeof(Vector4), hipMemcpyHostToDevice) );
}

APSS::~APSS()
{
    // device desallocation
    // ...
}

void APSS::select(const Vector3 &cameraPosition, const Vector3 &cameraDirection)
{
    checkVisibility<<<1,1>>>(m_sizeOriginal, m_positionOriginal, m_normalOriginal, cameraPosition, cameraDirection, m_visibility);
    CUDA_ASSERT( hipPeekAtLastError() );
    CUDA_ASSERT( hipDeviceSynchronize() );

    // prefix sum
    thrust::device_ptr<int> devPtr = thrust::device_pointer_cast(m_visibility);
    thrust::device_ptr<int> devPtrSum = thrust::device_pointer_cast(m_visibilitySum);
    thrust::exclusive_scan(thrust::device, devPtr, devPtr+m_sizeOriginal, devPtrSum, 0);

    updateSelectedCount();

    selectVisible<<<1,1>>>(m_sizeOriginal, m_visibility, m_visibilitySum, m_selected);
    CUDA_ASSERT( hipPeekAtLastError() );
    CUDA_ASSERT( hipDeviceSynchronize() );
}

void APSS::upsample(/*APSS parameters*/)
{
    //TODO
//    kernel<<</*numBlocks, blockSize*/>>>(...);
//    hipDeviceSynchronize();
}

void APSS::project(/*APSS parameters*/)
{
    //TEST : copy original in final
//    copy<<</*numBlocks, blockSize*/1,1>>>(m_sizeOriginal, m_positionOriginal, m_normalOriginal, m_colorOriginal,
//                                          m_sizeFinal,    m_positionFinal,    m_normalFinal,    m_colorFinal, m_splatSizeFinal);

    copySelected<<<1,1>>>(m_sizeSelected, m_positionOriginal, m_normalOriginal, m_colorOriginal,
                 m_selected, m_positionFinal, m_normalFinal, m_colorFinal, m_splatSizeFinal);

    m_sizeFinal = m_sizeSelected;

    CUDA_ASSERT( hipPeekAtLastError() );
    CUDA_ASSERT( hipDeviceSynchronize() );
}

void APSS::finalize()
{
    // get back final data from device to host
    CUDA_ASSERT( hipMemcpy(m_positionFinalHost,  m_positionFinal, m_sizeFinal*sizeof(Vector3), hipMemcpyDeviceToHost) );
    CUDA_ASSERT( hipMemcpy(m_normalFinalHost,    m_normalFinal, m_sizeFinal*sizeof(Vector3), hipMemcpyDeviceToHost) );
    CUDA_ASSERT( hipMemcpy(m_colorFinalHost,     m_colorFinal, m_sizeFinal*sizeof(Vector4), hipMemcpyDeviceToHost) );
    CUDA_ASSERT( hipMemcpy(m_splatSizeFinalHost, m_splatSizeFinal, m_sizeFinal*sizeof(Scalar), hipMemcpyDeviceToHost) );
}

void APSS::updateSelectedCount()
{
    m_sizeSelected = -1;
    CUDA_ASSERT( hipMemcpy(&m_sizeSelected, m_visibilitySum+m_sizeOriginal-1, sizeof(int), hipMemcpyDeviceToHost) );
}


} // namespace Cuda

} // namespace PointyCloudPlugin
